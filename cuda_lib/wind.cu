#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <chrono>
#include <memory>
#include "info.cuh"



std::chrono::time_point<std::chrono::high_resolution_clock> now() {
    return std::chrono::high_resolution_clock::now();
}

template <typename T>
double milliseconds(T t) {
    return (double) std::chrono::duration_cast<std::chrono::nanoseconds>(t).count() / 1000000;
}

__device__ double getVel(int x, int y, int z, double* u, int resolution, int dim);
__device__ double safe_get(int x, int y, int z, double* u, int resolution);
__global__ void jacobi(double* x_next, double* A, double* x_now, double* b, int* xyz, int Ni, int Res, int segment);


__global__
void bouyancyKernel(double* grid_temp, double* grid_q_v, double* grid_h, double* su_xyz,
                    int resolution, double dt)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < resolution*resolution*resolution && idx>-1)
    {
        double* src_u = su_xyz + idx*3;

        double T_th = (grid_temp[idx]-2)*10+20;
        double X_v = grid_q_v[idx]/(1+grid_q_v[idx]);
        double M_th = 18.02*X_v + 28.96*(1-X_v);
        double T_air = 20-0.0065*(grid_h[idx] + 20)*10;
        double buoyancy =   0.05*(28.96*T_th/(M_th*T_air) - 1);

        src_u[1] += buoyancy*dt;
    }
}

__global__
void advectKernel(double* su_xyz, int* id_xyz, double* tu_xyz,
                  int resolution, double cell_size, double dt)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < resolution*resolution*resolution && idx>-1)
    {
        int* this_xyz = id_xyz + idx*3;
        double* src_u = su_xyz + idx*3;
        double* dst_u = tu_xyz + idx*3;
        int x=this_xyz[0], y=this_xyz[1], z=this_xyz[2];
        double ua, ub;

        ua = getVel(x-1, y, z, su_xyz, resolution, 0);
        ub = getVel(x,   y, z, su_xyz, resolution, 0);
        dst_u[0] = src_u[0] - (ub - ua)/cell_size*(ub+ua)/2*dt;
        ua = getVel(x, y-1, z, su_xyz, resolution, 1);
        ub = getVel(x, y,   z, su_xyz, resolution, 1);
        dst_u[1] = src_u[1] - (ub - ua)/cell_size*(ub+ua)/2*dt;
        ua = getVel(x, y, z-1, su_xyz, resolution, 2);
        ub = getVel(x, y, z,   su_xyz, resolution, 2);
        dst_u[2] = src_u[2] - (ub - ua)/cell_size*(ub+ua)/2*dt;
    }
}

__global__
void viscosityKernel(double* su_xyz, int* id_xyz, double* tu_xyz, double viscosity,
                     int resolution, double cell_size, double dt)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < resolution*resolution*resolution && idx>-1)
    {
        int* this_xyz = id_xyz + idx*3;
        double* src_u = su_xyz + idx*3;
        double* dst_u = tu_xyz + idx*3;
        int x=this_xyz[0], y=this_xyz[1], z=this_xyz[2];

        if(x<resolution-1 && y<resolution-1 && z<resolution-1)
        {
            double factor = viscosity*dt/cell_size/cell_size;
            double u0, u1;
            u1 = getVel(x+1, y, z, su_xyz, resolution, 0);
            u0 = getVel(x-1, y, z, su_xyz, resolution, 0);
            dst_u[0] = src_u[0] + ((u1 - src_u[0]) - (src_u[0] - u0))*factor;

            u1 = getVel(x, y+1, z, su_xyz, resolution, 1);
            u0 = getVel(x, y-1, z, su_xyz, resolution, 1);
            dst_u[1] = src_u[1] + ((u1 - src_u[1]) - (src_u[1] - u0))*factor;

            u1 = getVel(x, y, z+1, su_xyz, resolution, 2);
            u0 = getVel(x, y, z-1, su_xyz, resolution, 2);
            dst_u[2] = src_u[2] + ((u1 - src_u[2]) - (src_u[2] - u0))*factor;
        }
        else
        {
            dst_u[0] = 0; dst_u[1] = 0; dst_u[2] = 0;
        }
    }
}

__global__
void pre_vorticityKernel(double* su_xyz, int* id_xyz, double* vorticity, double* vorticity_len,
                         int resolution, double cell_size, double dt)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < resolution*resolution*resolution && idx>-1)
    {
        int* this_xyz = id_xyz + idx*3;
        double* dst_v = vorticity + idx*3;
        double* dst_vL = vorticity_len + idx;

        int x=this_xyz[0], y=this_xyz[1], z=this_xyz[2];

        double tmp, len = 0;
        double grad0, grad1;
        grad0 = (getVel(x, y, z+1, su_xyz, resolution, 1) + getVel(x, y-1, z+1, su_xyz, resolution, 1) -
                 getVel(x, y, z-1, su_xyz, resolution, 1) - getVel(x, y-1, z-1, su_xyz, resolution, 1)); // grad_uy.z
        grad1 = (getVel(x, y+1, z, su_xyz, resolution, 1) + getVel(x, y+1, z-1, su_xyz, resolution, 1) -
                 getVel(x, y-1, z, su_xyz, resolution, 1) - getVel(x, y-1, z-1, su_xyz, resolution, 1)); // grad_uz.y
        tmp = (grad1 - grad0)/2/cell_size;
        dst_v[0] = tmp;
        len += tmp*tmp;

        grad0 = (getVel(x+1, y, z, su_xyz, resolution, 1) + getVel(x+1, y, z-1, su_xyz, resolution, 1) -
                 getVel(x-1, y, z, su_xyz, resolution, 1) - getVel(x-1, y, z-1, su_xyz, resolution, 1)); // grad_uz.x
        grad1 = (getVel(x, y, z+1, su_xyz, resolution, 1) + getVel(x-1, y, z+1, su_xyz, resolution, 1) -
                 getVel(x, y, z-1, su_xyz, resolution, 1) - getVel(x-1, y, z-1, su_xyz, resolution, 1)); // grad_ux.z
        tmp = (grad1 - grad0)/2/cell_size;
        dst_v[1] = tmp;
        len += tmp*tmp;

        grad0 = (getVel(x, y+1, z, su_xyz, resolution, 1) + getVel(x-1, y+1, z, su_xyz, resolution, 1) -
                 getVel(x, y-1, z, su_xyz, resolution, 1) - getVel(x-1, y-1, z, su_xyz, resolution, 1)); // grad_ux.y
        grad1 = (getVel(x+1, y, z, su_xyz, resolution, 1) + getVel(x+1, y-1, z, su_xyz, resolution, 1) -
                 getVel(x-1, y, z, su_xyz, resolution, 1) - getVel(x-1, y-1, z, su_xyz, resolution, 1)); // grad_uy.x
        tmp = (grad1 - grad0)/2/cell_size;
        dst_v[2] = tmp;
        len += tmp*tmp;

        dst_vL[0] = sqrt(len);
    }
}

__global__
void vorticityKernel(double* su_xyz, int* id_xyz, double* tu_xyz, double* vorticity, double* vorticity_len,
                     int resolution, double cell_size, double dt)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < resolution*resolution*resolution && idx>-1)
    {
        int* this_xyz = id_xyz + idx*3;
        double* src_u = su_xyz + idx*3;
        double* dst_u = tu_xyz + idx*3;
        int x=this_xyz[0], y=this_xyz[1], z=this_xyz[2];

        double vor_x = *(vorticity + idx*3);
        double vor_y = *(vorticity + idx*3 + 1);
        double vor_z = *(vorticity + idx*3 + 2);
        double dvor_x = (safe_get(x+1, y, z, vorticity_len, resolution) -
                         safe_get(x-1, y, z, vorticity_len, resolution))/2/cell_size;
        double dvor_y = (safe_get(x, y+1, z, vorticity_len, resolution) -
                         safe_get(x, y-1, z, vorticity_len, resolution))/2/cell_size;
        double dvor_z = (safe_get(x+1, y, z, vorticity_len, resolution) -
                         safe_get(x-1, y, z, vorticity_len, resolution))/2/cell_size;
        double len = sqrt(dvor_x*dvor_x+dvor_y*dvor_y+dvor_z*dvor_z) + 0.000001;
        dvor_x /= len; dvor_y /= len; dvor_z /= len;

        dst_u[0] = src_u[0] + (dvor_y*vor_z - dvor_z*vor_y)*cell_size*dt*0.01;
        dst_u[1] = src_u[1] + (dvor_z*vor_x - dvor_x*vor_z)*cell_size*dt*0.01;
        dst_u[2] = src_u[2] + (dvor_x*vor_y - dvor_y*vor_x)*cell_size*dt*0.01;
    }
}

__global__
void pre_JacobiKernel(double* su_xyz, int* id_xyz, double density_term, int resolution, double cell_size,
                     double* diag_A, double* rhs)
{

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < resolution*resolution*resolution && idx>-1)
    {
        int* this_xyz = id_xyz + idx*3;
        int x=this_xyz[0], y=this_xyz[1], z=this_xyz[2];
        double ua, ub;
        double this_rhs = 0;

        ua = getVel(x-1, y, z, su_xyz, resolution, 0);
        ub = getVel(x,   y, z, su_xyz, resolution, 0);
        this_rhs -= (ub - ua)/cell_size;
        ua = getVel(x, y-1, z, su_xyz, resolution, 1);
        ub = getVel(x, y,   z, su_xyz, resolution, 1);
        this_rhs -= (ub - ua)/cell_size;
        ua = getVel(x, y, z-1, su_xyz, resolution, 2);
        ub = getVel(x, y, z,   su_xyz, resolution, 2);
        this_rhs -= (ub - ua)/cell_size;
        rhs[idx] = this_rhs/density_term;

        double diag = 6;
        if(x>resolution-2) diag --;
        if(x<1) diag --;
        if(y>resolution-2) diag --;
        if(y<1) diag --;
        if(z>resolution-2) diag --;
        if(z<1) diag --;

        diag_A[idx] = diag;
    }
}

__global__
void pressureKernel(double* su_xyz, int* id_xyz, double* tu_xyz, double* pressure, double density,
                    int resolution, double cell_size, double dt)
{

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < resolution*resolution*resolution && idx>-1)
    {
        int* this_xyz = id_xyz + idx*3;
        double* src_u = su_xyz + idx*3;
        double* dst_u = tu_xyz + idx*3;
        int x=this_xyz[0], y=this_xyz[1], z=this_xyz[2];
        double dP;

        int face_num = resolution*resolution;
        if(x<resolution-1) dP = safe_get(x+1,y,z,pressure,resolution) - safe_get(x,y,z,pressure,resolution);
        else dP = 0;
        dst_u[0] = src_u[0] - dP*dt/(cell_size*density);
        if(y<resolution-1) dP = safe_get(x,y+1,z,pressure,resolution) - safe_get(x,y,z,pressure,resolution);
        else dP = 0;
        dst_u[1] = src_u[1] - dP*dt/(cell_size*density);
        if(z<resolution-1) dP = safe_get(x,y,z+1,pressure,resolution) - safe_get(x,y,z,pressure,resolution);
        else dP = 0;
        dst_u[2] = src_u[2] - dP*dt/(cell_size*density);
        if(x==0 || y==0 || z==0 || x==resolution-1 || y==resolution-1 || y==resolution-1)
        {
            dst_u[0] = 0; dst_u[1] = 0; dst_u[2] = 0;
        }
    }
}


extern "C"
void processWindGPU(double* grid_temp, double* grid_q_v, double* grid_h,
                    double* u_xyz, int* id_xyz, int jacobi_iter,
                    int resolution, double cell_size, float dt)
{
    double air_density = 1.225;
    double viscosity = 0.1;
    hipError_t err;

    auto t1 = now();
    int cell_num = resolution*resolution*resolution;
    double *d_temp, *d_q_v, *d_h, *d_u, *d_u2;
    int *d_id;
    hipMalloc(&d_temp, cell_num * sizeof(double)); // temperature
    hipMalloc(&d_q_v,  cell_num * sizeof(double)); // q_v
    hipMalloc(&d_h,    cell_num * sizeof(double)); // height
    hipMalloc(&d_u,    cell_num * sizeof(double) * 3); // vel 1
    hipMalloc(&d_u2,   cell_num * sizeof(double) * 3); // vel 2(for switching values)
    hipMalloc(&d_id,   cell_num * sizeof(int) * 3); // temperature

    hipMemcpy(d_temp,  grid_temp, cell_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_q_v,   grid_q_v,  cell_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_h,     grid_h,    cell_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_u,     u_xyz,     cell_num * sizeof(double) * 3, hipMemcpyHostToDevice);
    hipMemcpy(d_u2,    u_xyz,     cell_num * sizeof(double) * 3, hipMemcpyHostToDevice);
    hipMemcpy(d_id,    id_xyz,    cell_num * sizeof(int) * 3, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (cell_num - 1) / blockSize + 1;
    // advection
    auto t2 = now();
    advectKernel <<<numBlocks, blockSize>>> (d_u, d_id, d_u2, resolution, cell_size, dt);
    hipDeviceSynchronize();
//     diffusion
    auto t3 = now();
    viscosityKernel <<<numBlocks, blockSize>>> (d_u2, d_id, d_u, viscosity, resolution, cell_size, dt);
    hipDeviceSynchronize();
    // vorticity confinement
    auto t4 = now();
//    double *vorticity, *vorticity_len;
//    hipMalloc(&vorticity,      cell_num * sizeof(double) * 3);
//    hipMalloc(&vorticity_len,  cell_num * sizeof(double));
//    pre_vorticityKernel <<<numBlocks, blockSize>>> (d_u, d_id, vorticity, vorticity_len, resolution, cell_size, dt);
//    hipDeviceSynchronize();
//    vorticityKernel <<<numBlocks, blockSize>>> (d_u, d_id, d_u2, vorticity, vorticity_len, resolution, cell_size, dt);
//    hipDeviceSynchronize();
//    hipFree(vorticity);  hipFree(vorticity_len);
    // buoyancy
    auto t5 = now();
    bouyancyKernel <<<numBlocks, blockSize>>> (d_temp, d_q_v, d_h, d_u2, resolution, dt);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
                 __FILE__, __LINE__, hipGetErrorString( err) );
    }
    // pressure projection
    auto t6 = now();
    double *d_diag, *d_rhs = d_u, *x_now_d, *x_next_d;
    hipMalloc(&d_diag,   cell_num * sizeof(double));
    hipMalloc(&x_now_d,  cell_num * sizeof(double));
    hipMalloc(&x_next_d, cell_num * sizeof(double));
    hipMemset(x_now_d,  0, cell_num * sizeof(double));
    hipMemset(x_next_d, 0, cell_num * sizeof(double));
    //// jacoby iteration
    double density_term = dt/(air_density*cell_size*cell_size);
    pre_JacobiKernel<<<numBlocks, blockSize>>>(d_u2, d_id, density_term, resolution, cell_size, d_diag, d_rhs);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
                 __FILE__, __LINE__, hipGetErrorString( err) );
    }
    for (int k=0; k<jacobi_iter; k++)
    {
        if (k%2)
            jacobi <<<numBlocks, blockSize>>> (x_now_d, d_diag, x_next_d,
                                                 d_rhs, d_id, cell_num, resolution, 0);
        else
            jacobi <<<numBlocks, blockSize>>> (x_next_d, d_diag, x_now_d,
                                                 d_rhs, d_id, cell_num, resolution, 0);
        hipDeviceSynchronize();
    }
    double* pressure = x_next_d;
    //// apply pressure
    pressureKernel<<<numBlocks, blockSize>>>(d_u2, d_id, d_u, pressure, air_density,
                                             resolution, cell_size, dt);
    hipDeviceSynchronize();
    auto t7 = now();

    hipFree(d_diag);
    hipFree(x_now_d);
    hipFree(x_next_d);
    hipMemcpy(u_xyz, d_u, cell_num * sizeof(double) * 3, hipMemcpyDeviceToHost);

    hipFree(d_temp);
    hipFree(d_q_v);
    hipFree(d_h);
    hipFree(d_u);
    hipFree(d_u2);
    hipFree(d_id);

//    hipError_t err;
    err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
                 __FILE__, __LINE__, hipGetErrorString( err) );
    }

    std::cout << "[Wind Update Ellapse Summary]";
    std::cout << "-[Total- " << milliseconds(t7 - t1) << "]\n";
    std::cout << "[load- " << milliseconds(t2 - t1) << "]-";
    std::cout << "[advect- " << milliseconds(t3 - t2) << "]-";
    std::cout << "[diffuse- " << milliseconds(t4 - t3) << "]-";
    std::cout << "[vorticity- " << milliseconds(t5 - t4) << "]-";
    std::cout << "[buoyancy- " << milliseconds(t6 - t5) << "]-";
    std::cout << "[pressure- " << milliseconds(t7 - t6) << "]\n";
    std::cout << std::flush;
}



// Optimized device version of the Jacobi method
__global__ void jacobi(double* x_next, double* A, double* x_now, double* b, int* xyz, int Ni, int Res, int segment)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < Ni && idx>-1)
    {
        double sigma = 0.0;
        int id_x = xyz[idx*3];
        int id_y = xyz[idx*3+1];
        int id_z = xyz[idx*3+2];

        if(id_x > 0)     sigma += x_now[idx - Res*Res];
        if(id_x < Res-1) sigma += x_now[idx + Res*Res];
        if(id_y > 0)     sigma += x_now[idx - Res];
        if(id_y < Res-1) sigma += x_now[idx + Res];
        if(id_z > 0)     sigma += x_now[idx - 1];
        if(id_z < Res-1) sigma += x_now[idx + 1];

        x_next[idx] = (b[idx] + sigma) / A[idx];
//        printf(" (%d: %d, %d, %d) ", idx, id_x, id_y, id_z);
    }
}


__device__ double getVel(int x, int y, int z, double* u, int resolution, int dim)
{
    if(x<0 || y<0 || z<0 || x>resolution-1 || y>resolution-1 || y>resolution-1)
        return 0;
    if(dim==0 && x==resolution-1) return 0;
    if(dim==1 && y==resolution-1) return 0;
    if(dim==2 && z==resolution-1) return 0;
    int index = x*resolution*resolution + y*resolution + z;
    return u[index*3 + dim];
}

__device__ double safe_get(int x, int y, int z, double* u, int resolution)
{
    if(x<0 || y<0 || z<0 || x>resolution-1 || y>resolution-1 || y>resolution-1)
        return 0;
    int index = x*resolution*resolution + y*resolution + z;
    return u[index];
}
