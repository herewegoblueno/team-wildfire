#include<stdio.h>
#include<stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <chrono>
#include <memory>

std::chrono::time_point<std::chrono::high_resolution_clock> now() {
    return std::chrono::high_resolution_clock::now();
}

template <typename T>
double milliseconds(T t) {
    return (double) std::chrono::duration_cast<std::chrono::nanoseconds>(t).count() / 1000000;
}


// Optimized device version of the Jacobi method
__global__ void jacobi(double* x_next, double* A, double* x_now, double* b, int* xyz, int Ni, int Res, int segment)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < Ni && idx>-1)
    {
        double sigma = 0.0;
        int id_x = xyz[idx*3];
        int id_y = xyz[idx*3+1];
        int id_z = xyz[idx*3+2];

        if(id_x > 0)     sigma += x_now[idx - Res*Res];
        if(id_x < Res-1) sigma += x_now[idx + Res*Res];
        if(id_y > 0)     sigma += x_now[idx - Res];
        if(id_y < Res-1) sigma += x_now[idx + Res];
        if(id_z > 0)     sigma += x_now[idx - 1];
        if(id_z < Res-1) sigma += x_now[idx + 1];

        x_next[idx] = (b[idx] + sigma) / A[idx];
//        printf(" (%d: %d, %d, %d) ", idx, id_x, id_y, id_z);
    }
}

extern "C" void jacobiGPU(double* diag, double* rhs, int* id_xyz, int Res, int Ni, int iter)
{

    auto t1 = now();

    double* test_space;

    double *x_next_d, *diag_d, *x_now_d, *rhs_d;
    int* xyz_d;
    // Allocate memory on the device
    /*  Allocate Data  */

    hipMalloc((void **) &x_next_d, Ni*sizeof(double));
    hipMalloc((void **) &diag_d, Ni*sizeof(double))  ;
    hipMalloc((void **) &x_now_d, Ni*sizeof(double)) ;
    hipMalloc((void **) &rhs_d, Ni*sizeof(double))   ;
    hipMalloc((void **) &xyz_d, Ni*sizeof(int)*3)    ;


    // Copy data -> device
    hipMemcpy(diag_d, diag, sizeof(double)*Ni, hipMemcpyHostToDevice);
    hipMemcpy(rhs_d, rhs, sizeof(double)*Ni, hipMemcpyHostToDevice);
    hipMemset(x_next_d, 0, sizeof(double)*Ni);
    hipMemset(x_now_d, 0, sizeof(double)*Ni);
    hipMemcpy(xyz_d, id_xyz, sizeof(int)*Ni*3, hipMemcpyHostToDevice);


    auto t2 = now();


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int blockSize = 512;
    int numBlocks = (Ni - 1)/blockSize + 1;
    int totalThreads = blockSize*numBlocks;
    int segments = (Ni - 1)/totalThreads + 1;

    std::cout << "Threads per block:" << blockSize <<"\n Number of block:"  << numBlocks << "\n";
    for (int k=0; k<iter; k++)
    {
        if (k%2)
            jacobi <<< numBlocks, blockSize >>> (x_now_d, diag_d, x_next_d,
                                                 rhs_d, xyz_d, Ni, Res, 0);
        else
            jacobi <<< numBlocks, blockSize >>> (x_next_d, diag_d, x_now_d,
                                                 rhs_d, xyz_d, Ni, Res, 0);
        hipDeviceSynchronize();
//        for (int s=0;s<segments;s++)
//        {
//            int curent_max = (s+1)*totalThreads;
//            if(curent_max>Ni) curent_max = Ni;

//            int array_offset = s*totalThreads;
//            if(array_offset>0)
//            if (k%2)
//                jacobi <<< numBlocks, blockSize >>> (x_now_d, diag_d, x_next_d,
//                                                     rhs_d, xyz_d, curent_max, Res, s*totalThreads);
//            else
//                jacobi <<< numBlocks, blockSize >>> (x_next_d, diag_d, x_now_d,
//                                                     rhs_d, xyz_d, curent_max, Res, s*totalThreads);
//            cudaDeviceSynchronize();
//        }

    }


    hipError_t err;
    err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
                 __FILE__, __LINE__, hipGetErrorString( err) );
    }


    hipMemcpy(rhs, x_next_d, sizeof(double)*Ni, hipMemcpyDeviceToHost);
    hipFree(x_next_d); hipFree(diag_d); hipFree(x_now_d); hipFree(rhs_d); hipFree(xyz_d);

    auto t3 = now();

    std::cout << "[Jacobi Iteration]\n";
    std::cout << "Loading cost " << milliseconds(t2 - t1) << " milliseconds\n";
    std::cout << iter << " jacobi iterations cost " << milliseconds(t3 - t2) << " milliseconds\n";
    std::cout << std::flush;
}

////// for debug cuda mem check

//int main()
//{
//    int Res = 80;
//    int Ni = Res*Res*Res;
//    double* diag = (double*) malloc(sizeof(double)*Ni);
//    double* rhs = (double*) malloc(sizeof(double)*Ni);
//    int* id_xyz = (int*) malloc(3*sizeof(int)*Ni);

//    jacobiGPU(diag, rhs, id_xyz, Res, Ni, 20);
//    return 0;
//}
