
#include <hip/hip_runtime.h>


__global__
void windKernel(double* grid_temp, double* grid_q_v, double* grid_h) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    grid_q_v[index] = grid_q_v[index]/(1+grid_q_v[index]);
    grid_q_v[index] = 18.02*grid_q_v[index] + 28.96*(1-grid_q_v[index]);
    grid_h[index] = 7-0.0065*grid_h[index];
    grid_temp[index] = -0.1*(28.96*grid_temp[index]/grid_q_v[index]/grid_h[index] - 1);
}


extern "C"
void processWindGPU(double* grid_temp, double* grid_q_v, double* grid_h, int resolution) {

    double* cuda_temp;
    double* cuda_q_V;
    double* cuda_h;
    hipMalloc(&cuda_temp, resolution * sizeof(double));
    hipMalloc(&cuda_q_V, resolution * sizeof(double));
    hipMalloc(&cuda_h, resolution * sizeof(double));
    hipMemcpy(cuda_temp, grid_temp, resolution * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_q_V, grid_q_v, resolution * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_h, grid_h, resolution * sizeof(double), hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int blockSize = deviceProp.maxThreadsPerBlock;
    int numBlocks = (resolution - 1) / blockSize + 1;

    windKernel<<<numBlocks, blockSize>>>(cuda_temp, cuda_q_V, cuda_h);
    hipDeviceSynchronize();


    hipMemcpy(grid_temp, cuda_temp, resolution * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(cuda_temp);
    hipFree(cuda_q_V);


}
